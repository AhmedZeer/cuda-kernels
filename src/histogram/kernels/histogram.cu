#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include <torch/types.h>
#include <torch/extension.h>


// Binding Macroes.
#define STRING(val)  #val
#define BINDER(func) m.def(STRING(func), &func, STRING(func));

// Casting Macroes.
#define INT4(a) (reinterpret_cast<int4 *>(&(a))[0]);

// The items in array 'a' represents
// indices in 'b' array. We accumulate
// how many times an index in 'b' occurs
// in 'a' array, which is basically the definition
// of a histogram.

// Each thread is responsible of 1 element in 'a', and
// it will increment 'b' by +1. However, a bunch of threads
// could try to incerement 'b' at the same time, because
// different thread ids could correspond to the same index 
// in 'b' but NOT in 'a'.

// Histogram.
// 32-Bits
// blockDim(256), gridDim((N+256-1)/256)
// a: Nx1, b: histogramSize:1 
__global__ void histogram_i32_kernel(int *a, int *b, int N){
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(idx < N){
    atomicAdd(&b[a[idx]], 1);
  }
}

// Histogram.
// 32-Bits * 4 -> 128-Bits
// blockDim(256/4), gridDim((N+64-1)/64)
// a: Nx1, b: histogramSize:1 

// When we cast an arbitrary pointer at idx(i) to INT4,
// we get a[i:4] elements which we can directly use to
// construct 'int4' type element.
__global__ void histogram_i32x4_kernel(int *a, int *b, int N){
  int idx = 4 * (blockDim.x * blockIdx.x + threadIdx.x);

  if(idx + 3< N){
    int4 int4_a = INT4(a[idx]);
    atomicAdd(&b[a[int4_a.x]], 1);
    atomicAdd(&b[a[int4_a.y]], 1);
    atomicAdd(&b[a[int4_a.z]], 1);
    atomicAdd(&b[a[int4_a.w]], 1);
  }
}

#define CHECK_TENSOR_TYPE(a, dtype) \
    if((a).options().dtype() != (dtype)){ \
      throw std::runtime_error("Tensor dtypes doesnt match"); \
    }

#define CHECK_TENSOR_SIZE(a, size) \
    if((a).size(0) != (size)){ \
      throw std::runtime_error("Size Error"); \
    }

#define LAUNCHER(kernel_name, elm_per_thread, cast_type, tensor_type) \
  torch::Tensor histogram_##kernel_name##_launcher(torch::Tensor a){ \
    int N = a.size(0); \
    CHECK_TENSOR_TYPE(a, tensor_dtype) \
    CHECK_TENSOR_SIZE(a, N) \
    int BLOCKSIZE=256/elm_per_thread; \
    std::tuple<torch::Tensor, torch::Tensor> max_a = torch::max(a, 0); \
    int max_val = std::get<0>(max_a).cpu().item().to<int>(); \
    auto options = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA, 0); \
    auto b = torch::zeros({max_val+1}, options); \
    dim3 blockDim(BLOCKSIZE); \
    dim3 gridDim((256 + N - 1) / 256); \
    histogram_##kernel_name##_kernel<<<gridDim, blockDim>>> \
                      (reinterpret_cast<cast_type*>(a.data_ptr()), \
                      reinterpret_cast<cast_type*>(b.data_ptr()), \
                      N); \
    return b; \
  }


// Declare functions:
LAUNCHER(i32,   1, int, torch::kInt32);
LAUNCHER(i32x4, 4, int, torch::kInt32);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m){
  BINDER(histogram_i32_launcher)
  BINDER(histogram_i32x4_launcher)
}

