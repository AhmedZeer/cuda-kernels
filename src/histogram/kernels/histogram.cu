#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

// Binding Macroes.
#define STRINGIFY(str) #str
#define PYTORCH_MODULE(func) m.def(STRINGIFY(func), &func, STRINGIFY(func));

// Casting Macroes.
#define INT4(a) (reinterpert_cast<int4 *>(&(a))[0]);

// The items in array 'a' represents
// indices in 'b' array. We accumulate
// how many times an index in 'b' occurs
// in 'a' array, which is basically the definition
// of a histogram.

// Each thread is responsible of 1 element in 'a', and
// it will increment 'b' by +1. However, a bunch of threads
// could try to incerement 'b' at the same time, because
// different thread ids could correspond to the same index 
// in 'b' but NOT in 'a'.

// Histogram.
// 32-Bits
// blockDim(256), gridDim((N+256-1)/256)
// a: Nx1, b: histogramSize:1 
__global__ void histogram_i32_kernel(float *a, float *b, int N){
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(idx < N){
    atmoicAdd(&b[a[idx]], 1);
  }
}

// Histogram.
// 32-Bits * 4 -> 128-Bits
// blockDim(256), gridDim((N+256-1)/256)
// a: Nx1, b: histogramSize:1 

// When we cast an arbitrary pointer at idx(i) to INT4,
// we get a[i:4] elements which we can directly use to
// construct 'int4' type element.
__global__ void histogram_i32x4_kernel(float *a, float *b, int N){
  int idx = 4 * (blockDim.x * blockIdx.x + threadIdx.x);

  if(idx < N){
    int4 int4_a = INT4(a[idx]);
    atmoicAdd(&b[a[int4_a.x]], 1);
    atmoicAdd(&b[a[int4_a.y]], 1);
    atmoicAdd(&b[a[int4_a.z]], 1);
    atmoicAdd(&b[a[int4_a.w]], 1);
  }
}
