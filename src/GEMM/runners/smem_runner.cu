#include "../../utils/util.cuh"
#include "../headers/common.cuh"
#include "../kernels/SMEMCaching.cuh"
#include <stdio.h>

// Define 'uint' if not defined in common.cuh
#ifndef UINT_DEFINED
typedef unsigned int uint;
#define UINT_DEFINED
#endif

void runSMEMCaching(float *h_A, float *h_B, float *h_C_ref, uint m, uint n,
                    uint k) {
  // Host matrices
  float *h_C;
  float alpha = 1.0f, beta = 0.0f;
  const uint BLOCK_SIZE = 32; // Adjusted to 16 for better occupancy

  size_t size_A = m * k * sizeof(float);
  size_t size_B = k * n * sizeof(float);
  size_t size_C = m * n * sizeof(float);

  h_C = (float *)malloc(size_C);

  // Device matrices
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  // Copy data to device
  hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
  hipMemset(d_C, 0, size_C); // Initialize device C to zero

  // Define grid and block dimensions
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE); // 16x16 threads per block
  dim3 gridDim((n + BLOCK_SIZE - 1) / BLOCK_SIZE,
               (m + BLOCK_SIZE - 1) / BLOCK_SIZE);

  // Warmup loop
  for (int i = 0; i < 2; ++i) {
    SMEMCaching<BLOCK_SIZE>
        <<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, k, alpha, beta);
  }
  hipDeviceSynchronize(); // Ensure all operations are finished

  // Benchmark loop
  int numRuns = 3;
  float totalMilliseconds = 0.0f;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int i = 0; i < numRuns; ++i) {
    hipEventRecord(start);
    SMEMCaching<BLOCK_SIZE>
        <<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, k, alpha, beta);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    totalMilliseconds += milliseconds;
  }

  // Compute average execution time
  float averageMilliseconds = totalMilliseconds / numRuns;

  // Copy result back to host
  hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

  // Validate the result
  bool isValid = validateMatrices(h_C, h_C_ref, m, n, 1e-4f);
  printf("Validation: %s\n", isValid ? "SUCCESS" : "FAILURE");

  float maxDiff = maxDifferenceBetweenMatrices(h_C, h_C_ref, m, n);
  printf("Max Diff: %f\n", maxDiff);

  // Print performance metrics
  float seconds = averageMilliseconds / 1000.0f; // Convert to seconds
  float flop = 2.0f * m * n * k;           // FLOP for matrix multiplication
  float tflops = flop / (seconds * 1e12f); // TFLOPS
  float bandwidth = (size_A + size_B + size_C) / 1e9f / seconds; // GB/s

  printf("Kernel average execution time (ms): %f\n", averageMilliseconds);
  printf("Effective Bandwidth (GB/s): %f\n", bandwidth);
  printf("Performance (TFLOPS): %f\n", tflops);

  hipDeviceReset();
  // Clean up
  free(h_C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
