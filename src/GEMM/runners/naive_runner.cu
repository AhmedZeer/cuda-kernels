#include "hip/hip_runtime.h"
#include "../../utils/util.h"
#include "../headers/common.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_SIZE 256

// Kernel declaration from naive.cu
extern __global__ void naiveGEMM(float *A, float *B, float *C, int m, int n,
                                 int k);

void runNaiveGEMM(int m, int n, int k) {
  // Host matrices
  float *h_A, *h_B, *h_C, *h_C_ref;
  size_t size_A = m * k * sizeof(float);
  size_t size_B = k * n * sizeof(float);
  size_t size_C = m * n * sizeof(float);

  // Allocate host memory
  h_A = (float *)malloc(size_A);
  h_B = (float *)malloc(size_B);
  h_C = (float *)malloc(size_C);
  h_C_ref = (float *)malloc(size_C);

  // Initialize matrices
  initRandMatrix(h_A, m, k);
  initRandMatrix(h_B, k, n);

  // Perform CPU matrix multiplication for reference
  cpuMatmul(h_A, h_B, h_C_ref, m, n, k);

  // Device matrices
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_B, size_B);
  hipMalloc((void **)&d_C, size_C);

  // Copy data to device
  hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

  // Define grid and block dimensions
  int blockDim(BLOCK_SIZE); // 16x16 threads per block
  dim3 gridDim((n + blockDim - 1) / blockDim,
              (m + blockDim - 1) / blockDim);

  // Benchmark the kernel
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  naiveGEMM<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, k);
  hipEventRecord(stop);

  // Wait for the kernel to finish and measure time
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Copy result back to host
  hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

  // Validate the result
  bool isValid = validateMatrices(h_C, h_C_ref, m, n, 1e-4);
  printf("Validation: %s\n", isValid ? "SUCCESS" : "FAILURE");

  // Print performance metrics
  printf("Kernel execution time: %f ms\n", milliseconds);

  // Clean up
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_ref);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
